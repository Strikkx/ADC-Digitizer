#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <windows.h>
#include <time.h>


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 512


// Generate random float numbers
float* generateData(int n) {
    float* darr = (float*)malloc(sizeof(float) * n);

    srand(time(0));
    for (int i = 0; i < n; i++) {
        darr[i] = ((float)rand() / RAND_MAX) * 1000.0f;  // Generate random float between 0 and 1000
    }

    return darr;
}


__global__ void subtractionKernel(const float* data, int size, float* result) {
    __shared__ float shared_data_sub[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i + 3 < size) {
        shared_data_sub[tid] = (data[i] - data[i + 2]);
    }
    else {
        shared_data_sub[tid] = 0.0f;
    }

    __syncthreads();

    // Warp-level reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data_sub[tid] += shared_data_sub[tid + s];
        }
        __syncthreads();
    }

    // Write the result for each block to global memory
    if (tid == 0) {
        result[blockIdx.x] = shared_data_sub[0];
    }

}__global__ void multiplicationKernel(const float* data, int size, float* result) {
    __shared__ float shared_data_mult[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i + 3 < size) {
        shared_data_mult[tid] = (data[i] - data[i + 2]) * (data[i + 1] - data[i + 3]);
    }
    else {
        shared_data_mult[tid] = 0.0f;
    }

    __syncthreads();

    // Warp-level reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data_mult[tid] += shared_data_mult[tid + s];
        }
        __syncthreads();
    }

    // Write the result for each block to global memory
    if (tid == 0) {
        result[blockIdx.x] = shared_data_mult[0];
    }
}

__global__ void summationKernel(const float* data, int size, float* result) {
    __shared__ float shared_data_sum[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i + 3 < size) {
        shared_data_sum[tid] = (data[i] - data[i + 2]) * (data[i + 1] - data[i + 3]);
    }
    else {
        shared_data_sum[tid] = 0.0f;
    }

    __syncthreads();

    // Warp-level reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data_sum[tid] += shared_data_sum[tid + s];
        }
        __syncthreads();
    }

    // Write the result for each block to global memory
    if (tid == 0) {
        result[blockIdx.x] = shared_data_sum[0];
    }
}



int main() {

    // Generate random data
    const int n = 1000000;
    float* inputArray = generateData(n);
    size_t size = n * sizeof(float);

    // Calculate total blocks
    int blocks_sum = (n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE / 3;
    int blocks_sub = (n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE / 3;
    int blocks_mult = (n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE / 3;

    // Allocate GPU memory
    float* d_input_sum, * d_output_sum, * d_input_sub, * d_output_sub, * d_input_mult, * d_output_mult;
    hipMalloc((void**)&d_input_sum, size);
    hipMalloc((void**)&d_output_sum, blocks_sum * sizeof(float));

    hipMalloc((void**)&d_input_sub, size);
    hipMalloc((void**)&d_output_sub, blocks_sum * sizeof(float));

    hipMalloc((void**)&d_input_mult, size);
    hipMalloc((void**)&d_output_mult, blocks_sum * sizeof(float));

    // Calculate the time
    hipEvent_t start_sum, stop_sum, start_mult, stop_mult, start_sub, stop_sub;
    hipEventCreate(&start_sum);
    hipEventCreate(&stop_sum);

    hipEventCreate(&start_sub);
    hipEventCreate(&stop_sub);

    hipEventCreate(&start_mult);
    hipEventCreate(&stop_mult);

    // Copy data to device
    hipMemcpy(d_input_sum, inputArray, size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_sub, inputArray, size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_mult, inputArray, size, hipMemcpyHostToDevice);

    // Launch the kernel
    hipEventRecord(start_sum);
    summationKernel <<<blocks_sum, BLOCK_SIZE>>> (d_input_sum, size, d_output_sum);
    hipEventRecord(stop_sum);

    hipEventRecord(start_sub);
    subtractionKernel <<<blocks_sub, BLOCK_SIZE >>> (d_input_sub, size, d_output_sub);
    hipEventRecord(stop_sub);

    hipEventRecord(start_mult);
    multiplicationKernel <<<blocks_mult, BLOCK_SIZE >>> (d_input_mult, size, d_output_mult);
    hipEventRecord(stop_mult);


    // Copy the result from device to host
    float* result_sum = (float*)malloc(blocks_sum * sizeof(float));
    float* result_sub = (float*)malloc(blocks_sub * sizeof(float));
    float* result_mult = (float*)malloc(blocks_mult * sizeof(float));

    hipMemcpy(result_sum, d_output_sum, blocks_sum * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(result_sub, d_output_sub, blocks_sub * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(result_mult, d_output_mult, blocks_mult * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop_sum);
    hipEventSynchronize(stop_sub);
    hipEventSynchronize(stop_mult);


    float milliseconds_sum = 0;
    float milliseconds_sub = 0;
    float milliseconds_mult = 0;
    float milliseconds_total = 0;

    hipEventElapsedTime(&milliseconds_sum, start_sum, stop_sum);
    hipEventElapsedTime(&milliseconds_sub, start_sub, stop_sub);
    hipEventElapsedTime(&milliseconds_mult, start_mult, stop_mult);

    milliseconds_total = milliseconds_sum + milliseconds_sub + milliseconds_mult;

    // Sum the partial results from each block
    float final_result = 0.0f;
    for (int i = 0; i < blocks_sum; i++) {
        final_result += result_sum[i];
    }
    // Print time taken for each kernel
    printf("Summation took %f milliseconds to execute \n", milliseconds_sum);
    printf("Subtraction took %f milliseconds to execute \n", milliseconds_sub);
    printf("Multiplication took %f milliseconds to execute \n", milliseconds_mult);
    printf("All 3 kernels took %f milliseconds to execute \n", milliseconds_total);


    printf("Final result: %f\n", final_result);

    // Free device memory
    hipFree(d_input_sum);
    hipFree(d_output_sum);
    free(result_sum);

    hipFree(d_input_sub);
    hipFree(d_output_sub);
    free(result_sub);

    hipFree(d_input_mult);
    hipFree(d_output_mult);
    free(result_mult);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
